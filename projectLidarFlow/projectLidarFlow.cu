#include "hip/hip_runtime.h"
/* function for projecting lidar points
 *
 */

#include "../common.h"

__global__ void CameraTransformKernel(const float* const tform,
									  const float* const cam,
									  const size_t imWidth,
									  const size_t imHeight,
									  const float* const xIn,
									  const float* const yIn,
									  const float* const zIn,
									  const float* const xDIn,
									  const float* const yDIn,
									  const float* const zDIn,
									  const size_t numPoints,
									  float* const xOut,
									  float* const yOut,
									  float* const xDOut,
									  float* const yDOut,
									  bool* const valid){

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i >= numPoints){
		return;
	}

	//transform points
	float x = xIn[i]*tform[0] + yIn[i]*tform[4] + zIn[i]*tform[8] + tform[12];
	float y = xIn[i]*tform[1] + yIn[i]*tform[5] + zIn[i]*tform[9] + tform[13];
	float z = xIn[i]*tform[2] + yIn[i]*tform[6] + zIn[i]*tform[10] + tform[14];
	float xD = xDIn[i]*tform[0] + yDIn[i]*tform[4] + zDIn[i]*tform[8] + tform[12];
	float yD = xDIn[i]*tform[1] + yDIn[i]*tform[5] + zDIn[i]*tform[9] + tform[13];
	float zD = xDIn[i]*tform[2] + yDIn[i]*tform[6] + zDIn[i]*tform[10] + tform[14];

	bool v = true;
	if((z > 0)){
		//apply projective camera matrix
		x = cam[0]*x + cam[3]*y + cam[6]*z + cam[9];
		y = cam[1]*x + cam[4]*y + cam[7]*z + cam[10];
		z = cam[2]*x + cam[5]*y + cam[8]*z + cam[11];
		xD = cam[0]*xD + cam[3]*yD + cam[6]*zD + cam[9];
		yD = cam[1]*xD + cam[4]*yD + cam[7]*zD + cam[10];
		zD = cam[2]*xD + cam[5]*yD + cam[8]*zD + cam[11];

		//pin point camera model
		y = y/z;
		x = x/z;
		yD = yD/zD;
		xD = xD/zD;

		if((x < 0) || (y < 0) || (x >= imWidth) || (y >= imHeight)){
			v = false;
		}
	}
	else{
		v = false;
	}

	//output points
	xOut[i] = x;
	yOut[i] = y;
	xDOut[i] = xD-x;
	yDOut[i] = yD-y;
	valid[i] = v;
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
//initialize the MathWorks GPU API.
    mxInitGPU();

    //read data
    mxGPUArray const * tformMat = mxGPUCreateFromMxArray(prhs[0]);
    mxGPUArray const * camMat = mxGPUCreateFromMxArray(prhs[1]);
    mxGPUArray const * pointsMat = mxGPUCreateFromMxArray(prhs[2]);
    size_t imWidth = ((uint32_T *) mxGetData(prhs[3]))[1];
    size_t imHeight = ((uint32_T *) mxGetData(prhs[3]))[0];
    size_t numPoints = mxGPUGetDimensions(pointsMat)[0];

	
    //get input pointers
    float* tformPtr = (float*)(mxGPUGetDataReadOnly(tformMat));
    float* camPtr = (float*)(mxGPUGetDataReadOnly(camMat));

    float* xInPtr = (float*)(mxGPUGetDataReadOnly(pointsMat));
	float* yInPtr = &(xInPtr[numPoints]);
    float* zInPtr = &(yInPtr[numPoints]);
	float* xDInPtr = &(zInPtr[numPoints]);
	float* yDInPtr = &(xDInPtr[numPoints]);
	float* zDInPtr = &(yDInPtr[numPoints]);
	
    //create output
	mwSize outSize[] = {numPoints,4};
    mxGPUArray* outMat = mxGPUCreateGPUArray(2, outSize, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	plhs[1] = mxGPUCreateMxArrayOnGPU(outMat);
    outSize[1] = 1;
    mxGPUArray* validMat = mxGPUCreateGPUArray(2, outSize, mxLOGICAL_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	plhs[0] = mxGPUCreateMxArrayOnGPU(validMat);

    float* xOutPtr = (float*)(mxGPUGetData(outMat));
	float* yOutPtr = &(xOutPtr[numPoints]);
	float* xDOutPtr = &(yOutPtr[numPoints]);
	float* yDOutPtr = &(xDOutPtr[numPoints]);
	bool* validPtr = (bool*)(mxGPUGetData(validMat));

    //run and get ouputs
	CameraTransformKernel<<<gridSize(numPoints), BLOCK_SIZE>>>(tformPtr, camPtr, imWidth, imHeight, xInPtr, yInPtr, zInPtr, xDInPtr, yDInPtr, zDInPtr, numPoints, xOutPtr, yOutPtr, xDOutPtr, yDOutPtr,validPtr);
	CudaCheckError();
	
    //destroy reference structures
    mxGPUDestroyGPUArray(tformMat);
    mxGPUDestroyGPUArray(camMat);
    mxGPUDestroyGPUArray(pointsMat);
    mxGPUDestroyGPUArray(outMat);
	mxGPUDestroyGPUArray(validMat);
}
